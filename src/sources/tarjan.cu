#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "headers/lib.h"
#include "tarjan.h"

#define THREADxBLOCKalongXorY 16

// M Matrice di adiacenza
// N numero righe e colonne matrice di adiacenza
__global__ void tarjanDevice(int *M, int N, int *disc, int *low, int *visited, int *tarjan_stack, int *control_stack, int* stack_member)
{

    // row e col sono gli indici riga e colonna per l'inizio della sottomatrice
    // ogni thread processa una sottomatrice
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Facciamo delle prove solo per il primo blocco, dovrebbe trovare una SCC
    if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    {

        int control_stack_depth = 0;
        int tarjan_stack_depth = 0;
        // index di tarjan
        int index = 0;
        int i = row;

        // push nodo iniziale sullo stack
        tarjan_stack[threadIdx.x * N + tarjan_stack_depth] = i; // indice di riga
        stack_member[i] = 1;
        tarjan_stack_depth++;
    
        // push nodo iniziale sullo stack
        control_stack[threadIdx.x * N + control_stack_depth] = i; // indice di riga
        control_stack_depth++;

        disc[i] = index;
        low[i] = index;
        index++;

        
        while (control_stack_depth > 0)
        {
            // while control stack is not empty

            // pop
            int node = control_stack[threadIdx.x * N + control_stack_depth - 1];
            printf("%d\n", node);
            
            for (int k = 0; k < 10; k++)
            {
                if (M[node * N + k] == 1)
                {
                    if (visited[k] == 0)
                    {
                        visited[k] = 1;
                        control_stack[threadIdx.x * N + control_stack_depth] = k; // indice di riga
                        control_stack_depth++;

                        tarjan_stack[threadIdx.x * N + tarjan_stack_depth] = k; // indice di riga
                        stack_member[k] = 1;
                        tarjan_stack_depth++;

                        disc[k] = index;
                        low[k] = index;
                        index++;
                    }else if(stack_member[k] == 1){
                    
                        low[node] = low[node] < low[k] ? low[node] : low[k];
                    }
                }
            }

            int top = control_stack[threadIdx.x * N + control_stack_depth - 1];
            control_stack_depth--;

            if(control_stack_depth != 0){
                top = control_stack[threadIdx.x * N + control_stack_depth - 1];
                low[top] = low[top] < low[node] ? low[top] : low[node];
            }
            if(low[node] == disc[node]){
                
                printf("Found SCC, %d\n", node);
            }
            
        }
        printf("END WHILE\n");
    }
}

void to_adj_matrix(int *M, int width)
{
    FILE *file = fopen("../data/graph103", "r");
    if (file == NULL)
    {
        fprintf(stderr, "Error opening file\n");
        exit(1);
    }

    char *buffer = (char *)malloc(sizeof(char) * 10000);

    char delim[] = " ";
    int src = 0, dst = 0;
    char *token;
    while (fgets(buffer, 10000, file) != NULL)
    {
        printf("buffer: %s", buffer);
        token = strtok(buffer, delim);
        printf("LOG: reading row: %s\n", token);

        token = strtok(NULL, delim);
        while (token != NULL)
        {
            sscanf(token, "%d", &dst);
            printf("LOG: read: %s\n", token);
            M[src * width + dst] = 1;
            token = strtok(NULL, delim);
        }
        src++;
    }

    free(buffer);
}

int main()
{

    int width = 103; // number of vertices

    int *M = (int *)malloc(width * width * sizeof(int));
    memset(M, 0, width * width * sizeof(int));

    to_adj_matrix(M, width);

    // stampa matrice

    for (int y = 0; y < width; y++)
    {
        for (int x = 0; x < width; x++)
        {
            printf("%d ", M[y * width + x]);
        }
        printf("\n");
    }

    // CUDA grid management
    int gridsize = width / THREADxBLOCKalongXorY;

    if (gridsize * THREADxBLOCKalongXorY < width)
    {
        gridsize = gridsize + 1;
    }

    dim3 dimGrid(gridsize, gridsize);
    dim3 dimBlock(THREADxBLOCKalongXorY, THREADxBLOCKalongXorY);
    printf("Gridsize: %d\n", gridsize);

    int *dev_M;
    int *dev_disc, *dev_low, *dev_visited;
    int *dev_control, *dev_tarjan, *dev_member;
    int size = width * width * sizeof(int);
    hipMalloc((void **)&dev_M, size);

    hipMalloc((void **)&dev_disc, width * sizeof(int));
    hipMalloc((void **)&dev_low, width * sizeof(int));
    hipMalloc((void **)&dev_visited, width * sizeof(int));
    hipMalloc((void **)&dev_control, width * sizeof(int));
    hipMalloc((void **)&dev_tarjan, width * sizeof(int));
    hipMalloc((void **)&dev_member, width * sizeof(int));

    hipMemcpy(dev_M, M, size, hipMemcpyHostToDevice);

    hipMemset(dev_disc, -1, width * sizeof(int));
    hipMemset(dev_low, -1, width * sizeof(int));
    hipMemset(dev_visited, 0, width * sizeof(int));
    hipMemset(dev_control, 0, width * sizeof(int));
    hipMemset(dev_tarjan, 0, width * sizeof(int));
    hipMemset(dev_member, 0, width * sizeof(int));


    hipError_t mycudaerror;
    mycudaerror = hipGetLastError();

    // kernel launch
    tarjanDevice<<<dimGrid, dimBlock>>>(dev_M, width, dev_disc, dev_low, dev_visited, dev_tarjan, dev_control, dev_member);
    mycudaerror = hipGetLastError();
    if (mycudaerror != hipSuccess)
    {
        fprintf(stderr, "%s\n", hipGetErrorString(mycudaerror));
        exit(1);
    }
    hipDeviceSynchronize();
    hipFree(dev_M);
    hipFree(dev_disc);
    hipFree(dev_low);
    free(M);
    printf("Terminated");
    exit(0);
}
